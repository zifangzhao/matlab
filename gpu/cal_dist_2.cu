
#include <hip/hip_runtime.h>

__global__ void cal_dist_2(double * dist,double * vecs,int * vref,int * tim_range,int embedding)
{

int tim_idx=threadIdx.x; //vecs location
//int tim_idx=blockIdx.y; //tim_range
//int tim_idx=gridDim.x; //tim_range
int ref_idx=blockIdx.y;//startpoint


int ebd;
int tim;
int refe;

double temp=0;
double len=0;

refe=vref[ref_idx]-1;   //matlab������1��ʼ��c��0��ʼ
tim=refe+tim_range[tim_idx];  //tim_range=w1:w2;

for(ebd=0;ebd<embedding;ebd++){
    len=vecs[tim*embedding+ebd]-vecs[refe*embedding+ebd];
    temp+=len*len;
}
dist[ref_idx+tim_idx*gridDim.y]=sqrt(temp);
}