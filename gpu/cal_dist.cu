
#include <hip/hip_runtime.h>

__global__ void cal_dist(double * dist,double * vecs,int embedding)
{

int idx=blockIdx.x * blockDim.x +threadIdx.x;
int i;
double temp=0;

for(i=0;i<embedding;i++){
    temp+=vecs[idx+i]*vecs[idx+i];
}
dist[idx]=sqrt(temp);
}