#include "hip/hip_runtime.h"
# pragma warning (disable:4819)
#include "mex.h"
// #include "hip/hip_runtime.h"
// #include "hip/hip_runtime.h"
#include "gpu/mxGPUArray.h"
#define BlockSize 1024
#define NPRCMAX 1000

//Device code 1 for single channel distance calculation
void __global__ CalDistance(double * dist,const double * rawdata,int N_data,int Nchn,const double * vref,const double * tim_range, int embedding,int sampling_delay,int Nvec,int Nstep){
/*���ڵõ�������Ϊһ��ʱ��������Ĳ���
 *���ݲ�ͬ��step��window��С���ж���Ҫ�����thread����
 *һ��step��Ҫ����w2-w1��������distance,��ͬһͨ���ڷ�Ϊstep��������channel���Ի�����block��
 *
 */

int t_idx=threadIdx.x; //this is the vector idx plus step idx
int b_idx=blockIdx.x;// the number of single channel's block idx
int chn_idx=blockIdx.y;//this is the channel idx

int tim_idx;//keep the same with cal_dist_3
int ref_idx;  //startpoint

// extern __shared__ double shared_data[]; //should be size of single channel rawdata;

//filling up the shared memory, this should have nothing to do with b_idx
int temp_idx=t_idx; // this is for filling the shared memory
// int t;
// while(temp_idx<N_data){
// //      t=rawdata[chn_idx+temp_idx*Nchn];
//     shared_data[temp_idx]=rawdata[chn_idx+temp_idx*Nchn]; //��1��channel�����ݷ��䵽�����ڴ���,all thread is involved to acceletate
//     temp_idx+=BlockSize;
//     
// }
// __syncthreads();
//Data transfer complete,data now transferd to shared memory

//get the actural cordinate  by tim_idx and ref_idx
temp_idx=t_idx+b_idx*BlockSize; //get the current abs cord.
if(temp_idx<Nvec*Nstep){ //��Ϊtheradsperblock�ǹ̶��ģ���˻����߳̿鲢��������ڼ��㣬ֻ���빲���ڴ����
    ref_idx=(int) temp_idx/(Nvec);//stp
    tim_idx=temp_idx % (Nvec);//vec_idx
// //------------------------------calculating the distance---------------------------------
    int tim;
    int refe;
    double temp=0;
    double len=0;
    
//�����������������ԭʼ�����еľ�������
    refe=vref[ref_idx];   //matlab������1��ʼ��c��0��ʼ !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    tim=refe+tim_range[tim_idx];  //tim_range=w1:w2;!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    
    for(int ebd=0;ebd<embedding;ebd++){
        len=rawdata[chn_idx+(refe+ebd*sampling_delay)*Nchn]-rawdata[chn_idx+(tim+ebd*sampling_delay)*Nchn];
        temp+=len*len;
//         temp=shared_data[N_data-1];//shared data OK!
//         temp=tim;//201-202????
    }
//     if(t_idx+b_idx*BlockSize<N_data){
//     dist[t_idx+b_idx*BlockSize]=shared_data[t_idx+b_idx*BlockSize];
//     }
    dist[ref_idx+tim_idx*Nstep+chn_idx*Nvec*Nstep]=temp;//shared memory would work through block
}
}
void __device__ gpu_sort(double *min_num,double add_num,int N_small){ //pick up smallest N_small values
//     int *temp=new temp[N_small+1];
    double max_v=add_num;
    double temp;
    
    
    for(int i=0;i<N_small;i++){
        if(min_num[i]>max_v){
            temp=min_num[i];
            min_num[i]=max_v;
            max_v=temp;
        }
    }    
}
double __device__ gpu_max(double * data, int N){
    double max_v=data[0];
    for(int i=1;i<N;i++){
        if(data[i]>max_v){
            max_v=data[i];
        }
    }
    return max_v;
}

void __global__ GPUprctile(double * dist,double * cdist,int N_small,int Nstep,int Nvec){
    int idx_stepA=threadIdx.x;
    int idx_stepB=blockIdx.x;
    int chn_idx=blockIdx.y;
    int stp_idx=idx_stepA+BlockSize*idx_stepB;
   double min_num[NPRCMAX];
   for(int i=0;i<N_small;i++){
       min_num[i]=1.79769e+308;
   }
    if(stp_idx<Nstep){
        for(int i=0;i<Nvec;i++){
            gpu_sort(min_num,dist[stp_idx+i*Nstep+chn_idx*Nvec*Nstep],N_small);
        }
        cdist[stp_idx+chn_idx*Nstep]=gpu_max(min_num,N_small);
    }
    __syncthreads();
    
}

/*---------------------------------------Convert distace into pattern-------------*/
void __global__ CalPattern(int * pattern, double * dist,double * cdist,int Nvec,int Nstep){
//-----------------------calculating pattern---------------------------
    int t_idx=threadIdx.x; //this is the vector idx plus step idx
    int b_idx=blockIdx.x;// the number of single channel's block idx
    int chn_idx=blockIdx.y;//this is the channel idx
    
    int tim_idx;//keep the same with cal_dist_3
    int ref_idx;

    int temp_idx=t_idx+b_idx*BlockSize; // get the current abs cord.
    if(temp_idx<Nvec*Nstep){
        ref_idx=(int) temp_idx/Nvec; //stp
        tim_idx=temp_idx % Nvec;//vec_idx
        
        pattern[ref_idx+tim_idx*Nstep+chn_idx*Nvec*Nstep]=(dist[ref_idx+tim_idx*Nstep+chn_idx*Nvec*Nstep]<=cdist[ref_idx+chn_idx*Nstep]);
    }
}
/*---------------------------------------Convert pattern into SL matrix-------------*/
void __global__ SLGen(double * SL, int * pattern,const double * vref,double Nvec,double Nstep,double Nchn,double Nstart){
    int tim_idx=threadIdx.x; //vecs location,this can not be longer than 1024
//     int step_idx=blockIdx.y; //���idx��startpoint��delay�Ļ��һά����Ҫ����
    int stp=blockIdx.y;
    int dly=blockIdx.z;
    int chn=blockIdx.x;// channel number A
//     int y=blockIdx.y;// channel number B
    int x=chn/(int)Nchn;
    int y=chn%(int)Nchn;
    int synE=0;
    int allE=0;
    __shared__ int syn[2*BlockSize+1]; //��Ҫ����ʼ���Ƿ�Ϊ0
    
//     int dly=step_idx/(int)Nstart;
//     int stp=step_idx%(int)Nstart;
    int A=pattern[(int)(stp+tim_idx*Nstep+x*Nvec*Nstep)];
    int B=pattern[(int)((stp+dly)+tim_idx*Nstep+y*Nvec*Nstep)];
//     int B=pattern[(int)(tim_idx+(stp+dly)*Nvec+y*Nvec*Nstep)];
    syn[2*tim_idx]=A&B;
    syn[1+2*tim_idx]=A+B;
    __syncthreads();
    if(tim_idx==0){
        for(int idx=0;idx<Nvec;idx++){
        synE+=syn[2*idx];
        allE   +=syn[1+2*idx];
        }
        if(allE!=0){
            SL[(int)(x+y*Nchn+stp*Nchn*Nchn+dly*Nstart*Nchn*Nchn)]=2*((double) synE/ (double) allE);
        }
        else{
            SL[(int)(x+y*Nchn+stp*Nchn*Nchn+dly*Nstart*Nchn*Nchn)]=0;
        }
    }
    __syncthreads();
}
/*
 * Host code----------------------------------------------MAIN FUNCTION--------------------------------------------
 -----------------------------------------------------------------------------------------------------------------------*/
void mexFunction(int nlhs,mxArray *plhs[],int nrhs,mxArray const * prhs[]){
    //(rawdata,w1,w2,embedding,sampling_delay,p_ref,stps,delays)
    //plhs=output,need mxCreateDoubleMatrix to assign the space
    
    int const threadsPerBlock=BlockSize;
    
     /* Declare all variables.*/
    mxGPUArray const *rawdata; 
    mxGPUArray * dist;
    mxGPUArray * pattern;
    mxGPUArray * SL;
    mxGPUArray * cdist;
    mxGPUArray const * GVref;
    mxGPUArray const * Gtim;
//     mxGPUArray * min_num;
    
    mxArray * vref;
    mxArray * tim;
    mxArray * para;
//     mxArray * ref_out;
//     mxArray * tim_out;
//     mxArray * stp_out;
    double * p_vref;
    double * p_tim;
    
    double const * p_rawdata;
    double * p_dist; 
    int * p_pattern;
    double * p_SL;
    double * p_cdist;
    double const * p_GVref;
    double const * p_Gtim;
//     double * p_min_num;
    
    int Nchn;
    int Datalen;
    int Nvec;
    int Nstep;
    
//     char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
//     char const * const errMsg = "Invalid input to MEX file.";
    
    //initialize the MATLAB gpu API
    mxInitGPU();
    /*--------------------------------testing input parameters-----------------------------*/
    /* Throw an error if the input is not a GPU array. */
//     if ((nrhs!=8) || !(mxIsGPUArray(prhs[0]))) {
//         mexErrMsgIdAndTxt(errId, errMsg);
//     }
    /*--------------------------------Creating mex vars----------------------------*/
    //(rawdata,w1,w2,embedding,sampling_delay,p_ref,stps,delays)
    Nchn=mxGetM(prhs[0]);
    Datalen=mxGetN(prhs[0]);
    rawdata=mxGPUCreateFromMxArray(prhs[0]); /*Create read-only mxGPUArray object from input mxArray
     * ???????????????????const��ԭ��
     *data transfered to GPUarray*/
    p_rawdata=(double const *)(mxGPUGetDataReadOnly(rawdata));//Read-only raw pointer to underlying data
   
    double w1=mxGetScalar(prhs[1]);
    double w2=mxGetScalar(prhs[2]);
    double embedding=mxGetScalar(prhs[3]);
    double sampling_delay=mxGetScalar(prhs[4]);
    double p_ref=mxGetScalar(prhs[5]);
    double * stps=mxGetPr(prhs[6]);
    double * delays=mxGetPr(prhs[7]);
    int Nstart=mxGetN(prhs[6]);
    int Ndelay=mxGetN(prhs[7]);
    Nvec=(int) (w2-w1-(embedding-1)*sampling_delay+1)*2/sampling_delay;
    Nstep=Nstart+Ndelay;
    int N_small=Nvec*p_ref;
    int temp_size[]={Nchn*Nvec*Nstep};
        /* Create a GPUArray to hold the result and get its underlying pointer. */               
    
    dist=mxGPUCreateGPUArray(1,
            (const mwSize *)temp_size,//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!��һ����ȷ
            mxDOUBLE_CLASS,
            mxREAL,
            MX_GPU_INITIALIZE_VALUES); //���һ�о����Ƿ��ʼ��
    p_dist=(double * )mxGPUGetData(dist); //???????????????Ϊʲô��Ҫǿ��ת������
    pattern=mxGPUCreateGPUArray(1,
            (const mwSize *)temp_size,
            mxINT32_CLASS, //_________________________________________caution
            mxREAL,
            MX_GPU_INITIALIZE_VALUES); //���һ�о����Ƿ��ʼ��
    p_pattern=(int * )mxGPUGetData(pattern);
    temp_size[0]=Nchn*Nchn*Nstart*Ndelay;
    SL=mxGPUCreateGPUArray(1,
            (const mwSize *)temp_size,
            mxDOUBLE_CLASS,
            mxREAL,
            MX_GPU_INITIALIZE_VALUES); //���һ�о����Ƿ��ʼ��
    p_SL=(double * )mxGPUGetData(SL);
    temp_size[0]=Nchn*Nstep;
    cdist=mxGPUCreateGPUArray(1,
            (const mwSize *)temp_size,//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!��һ����ȷ
            mxDOUBLE_CLASS,
            mxREAL,
            MX_GPU_INITIALIZE_VALUES); //���һ�о����Ƿ��ʼ��
    p_cdist=(double * )mxGPUGetData(cdist); 
//     temp_size[0]=N_small;
//     min_num=mxGPUCreateGPUArray(1,
//              (const mwSize *)temp_size,//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!��һ����ȷ
//             mxDOUBLE_CLASS,
//             mxREAL,
//             MX_GPU_INITIALIZE_VALUES); //���һ�о����Ƿ��ʼ��
//     p_min_num=(double * )mxGPUGetData(min_num); 
            

    
    /*--------------------------------establish output variables---------------------------*/
    plhs[0]=mxCreateCellMatrix(1,1);
    //nlhs doesn't count?
    
    /*--------------------------------main program--------------------------------------*/
    int blocksPerGrid=Nchn*(1+Nvec*Nstep/threadsPerBlock);
    

    
    int tim_size=2*(w2-w1-(embedding-1)*sampling_delay+1);
    vref=mxCreateDoubleMatrix(Nstep,1,mxREAL);
    tim=mxCreateDoubleMatrix(tim_size,1,mxREAL);
    p_vref=mxGetPr(vref);
    p_tim=mxGetPr(tim);
    
    
    //������Ҫ�����ʵ����Ҫ��step,������stp��dly���Ǵ�0��ʼ�����Ҳ���һ����stps����>1
    int inc=stps[1]-stps[0];
    for(int idx=0;idx<Nstep;idx++){
        p_vref[idx]=inc*(idx+stps[0])+w2;//location of Ref vector, should be stps+w2
    }
    
    for(int idx=0;idx<tim_size;idx++){ //[-w2:-w1-ebd-1)*sampling_delay,w1:w2-(ebd-1)*sampling_delay
        if(idx<tim_size/2){
            p_tim[idx]=idx-w2;
//             p_tim[idx]=idx*sampling_delay-w2;
        }
        else{
            p_tim[idx]=w1+idx-tim_size/2;
//             p_tim[idx]=w1+idx*sampling_delay-tim_size/2;
        }
    }
    
    
//     int *vref=new int[Nstart];
//     for(int idx=0;idx<Nstart;idx++){
//         vref[idx]=stps[idx]+w2;
//     }
//     int *tim=new int[tim_size];
//     for(int idx=0;idx<tim_size;idx++){
//         if(idx<tim_size/2){
//             tim[idx]=idx-w2;
//         }
//         else{
//             tim[idx]=idx+2*w1+(embedding-1)*sampling_delay;
//         }
//     }
    
    GVref=mxGPUCreateFromMxArray(vref);
    Gtim=mxGPUCreateFromMxArray(tim);
    p_GVref=(double const *)(mxGPUGetDataReadOnly(GVref));
    p_Gtim=(double const *)(mxGPUGetDataReadOnly(Gtim));
    
//     p_GVref=(double * )mxGPUGetData(GVref); 
//     p_Gtim=(double * )mxGPUGetData(Gtim); 
    
    dim3 blocks(1+Nvec*Nstep/threadsPerBlock,Nchn);
//    CalDistance<<<blocks,threadsPerBlock,Datalen*sizeof(double)>>>(p_dist,p_rawdata,Datalen,Nchn,p_GVref,p_Gtim,(int) embedding,(int) sampling_delay,Nvec,Nstep);
   CalDistance<<<blocks,threadsPerBlock>>>(p_dist,p_rawdata,Datalen,Nchn,p_GVref,p_Gtim,(int) embedding,(int) sampling_delay,Nvec,Nstep);
   mxGPUDestroyGPUArray(rawdata);
   
   
   dim3 blocks_C(1+Nstep/threadsPerBlock,Nchn);
   GPUprctile<<<blocks_C,threadsPerBlock>>>(p_dist,p_cdist,N_small,Nstep,Nvec); 
   //calculate the critical distance in CPU
   
   CalPattern<<<blocks,threadsPerBlock>>>(p_pattern,p_dist,p_cdist,Nvec,Nstep);
   mxGPUDestroyGPUArray(dist);
   mxGPUDestroyGPUArray(cdist);
   
   dim3 blocks_SL(Nchn*Nchn,Nstart,Ndelay);
   SLGen<<<blocks_SL,Nvec>>>(p_SL,p_pattern,p_GVref,Nvec,Nstep,Nchn,Nstart);
   mxGPUDestroyGPUArray(pattern);
    //dist[ref_idx+tim_idx*Nvec+chn_idx*Nvec*Nstep]
    //pattern[ref_idx+tim_idx*Nvec+chn_idx*Nvec*Nstep]
    //SL[x+y*Nchn]
    
    //convert the SL into CELL

    para=mxCreateDoubleMatrix(6,1,mxREAL);//����һ��Ƭ�ڴ棬�ṩָ���ַ��ָ��
//     ref_out=mxCreateDoubleMatrix(Nstart,1,mxREAL);
//     tim_out=mxCreateDoubleMatrix(tim_size,1,mxREAL);
//     stp_out=mxCreateDoubleMatrix(Nstart,1,mxREAL);
    double * para_C=mxGetPr(para);
//     double * para_CR=mxGetPr(ref_out);
//     double * para_CT=mxGetPr(tim_out);
//     double * para_stp=mxGetPr(stp_out);
    double temp[]={Datalen,Nchn,temp_size[0],tim_size,blocksPerGrid,threadsPerBlock};
//     para_C[0]=Datalen;
//     para_C[1]=Nchn;
//     para_C[2]=embedding;
//     para_C[3]=sampling_delay;
//     para_C[4]=Nvec;
//     para_C[5]=Nstep;
    memcpy(para_C,temp,sizeof(temp));

//     memcpy(para_CR,p_vref,Nstep*sizeof(double));
//     memcpy(para_CT,p_tim,tim_size*sizeof(double));
//     memcpy(para_stp,stps,Nstart*sizeof(double));
    

    mxSetCell(plhs[0],0,mxGPUCreateMxArrayOnCPU(SL));  //testing by output to 1
//     mxSetCell(plhs[0],1,mxGPUCreateMxArrayOnCPU(rawdata));  //testing by output to 1
//     mxSetCell(plhs[0],2,para);
//     mxSetCell(plhs[0],3,vref);
//     mxSetCell(plhs[0],4,tim);
//     mxSetCell(plhs[0],3,ref_out);
//     mxSetCell(plhs[0],4,tim_out);
//     mxSetCell(plhs[0],5,mxGPUCreateMxArrayOnCPU(dist));  //testing by output to 1
//     mxSetCell(plhs[0],6,mxGPUCreateMxArrayOnCPU(cdist));  //testing by output to 1
//     mxSetCell(plhs[0],7,mxGPUCreateMxArrayOnCPU(pattern));  //testing by output to 1
    
//     mxSetCell(plhs[0],8,stp_out);
  //  mxArray * SL_CPU=mxGPUCreateMxArrayOnCPU(SL);
    //mxArray * SL_each=mxCreateDoubleMatrix(Nstart,Ndelay,mxREAL);
//     double * p_SL_each=mxGetPr(SL_each);
//     double * p_SL_CPU=mxGetPr(SL_CPU);
//     for(int A=0;A<Nchn;A++){
//         for(int B=0;B<Nchn;B++){
//             for(int x=0;x<Nstart;x++){
//                 for(int y=0;y<Ndelay;y++){
//                     p_SL_each[x+y*Nstart]=p_SL_CPU[A+B*Nchn+x*Nchn*Nchn+y*Nchn*Nchn*Nstart];
//                 }
//             }
//             mxSetCell(plhs[0],8+A+B*Nchn,SL_each);
//         }        
//     }

    
    /*--------------------------------release resource------------------------------------*/
//     delete[] vref;
//     delete[] tim;
//     mxGPUDestroyGPUArray(dist);
//     mxGPUDestroyGPUArray(pattern);
    mxGPUDestroyGPUArray(SL);
//     mxGPUDestroyGPUArray(rawdata);    
//     mxGPUDestroyGPUArray(cdist);
    
//     mxDestroyArray(vref);
//     mxDestroyArray(tim);
//     mxDestroyArray(para);
//     mxDestroyArray(ref_out);
//     mxDestroyArray(tim_out);
//     mxDestroyArray(stp_out);

//     mxGPUDestroyGPUArray(min_num);    

    mxGPUDestroyGPUArray(GVref);    
    mxGPUDestroyGPUArray(Gtim);    
//     hipDeviceReset();
    return;
}