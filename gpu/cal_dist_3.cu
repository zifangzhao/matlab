
#include <hip/hip_runtime.h>

__global__ void cal_dist_3(double * dist,double * vecs,int * vref,int * tim_range,int embedding)
{

int tim_idx=threadIdx.x; //vecs location,��ÿһ�����������
//int tim_idx=blockIdx.y; //tim_range
//int tim_idx=gridDim.x; //tim_range
int ref_idx=blockIdx.y;//startpoint,�൱���ƶ���ref

extern __shared__ float vecs_shared[];
//__shared__ float vecs_ref_shared[blockDim.x];
int ebd;
int tim;
int refe;

double temp=0;
double len=0;

refe=vref[ref_idx]-1;   //matlab������1��ʼ��c��0��ʼ
tim=refe+tim_range[tim_idx];  //tim_range=w1:w2;



for(ebd=0;ebd<embedding;ebd++){
    vecs_shared[tim_idx*embedding+ebd]=vecs[tim*embedding+ebd];

    len=vecs_shared[tim_idx*embedding+ebd]-vecs[refe*embedding+ebd];
    temp+=len*len;
}
dist[ref_idx+tim_idx*gridDim.y]=sqrt(temp);
}