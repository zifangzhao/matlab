
#include <hip/hip_runtime.h>

__global__ void cal_dist_3(double * dist,double * vecs,int * vref,int * tim_range,int embedding)
{

int tim_idx=threadIdx.x; //vecs location,对每一个向量求距离
//int tim_idx=blockIdx.y; //tim_range
//int tim_idx=gridDim.x; //tim_range
int ref_idx=blockIdx.y;//startpoint,相当于移动的ref

extern __shared__ float vecs_shared[];
//__shared__ float vecs_ref_shared[blockDim.x];
int ebd;
int tim;
int refe;

double temp=0;
double len=0;

refe=vref[ref_idx]-1;   //matlab索引从1开始，c从0开始
tim=refe+tim_range[tim_idx];  //tim_range=w1:w2;



for(ebd=0;ebd<embedding;ebd++){
    vecs_shared[tim_idx*embedding+ebd]=vecs[tim*embedding+ebd];

    len=vecs_shared[tim_idx*embedding+ebd]-vecs[refe*embedding+ebd];
    temp+=len*len;
}
dist[ref_idx+tim_idx*gridDim.y]=sqrt(temp);
}