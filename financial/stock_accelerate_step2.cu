
#include <hip/hip_runtime.h>

__global__ void stock_accelerate_step2(double * score,double * data,double * post,int paras_seg,int paras,int days){

int day_idx=threadIdx.x; //day
//int para_idx=blockIdx.x; //parameters
int popu_idx=blockIdx.y; //population

double sum=0;
double post_temp=0;
double diff=0;
for (int para_idx=0;para_idx<paras;para_idx++){
  post_temp=0;
  for (int idx=0;idx<paras_seg;idx++){
     post_temp=post_temp+post[idx+(para_idx+(day_idx+(popu_idx)*days)*paras)*paras_seg];
   }//post[para_idx1,para_idx,day_idx,popu_idx]
  diff=(post_temp-data[para_idx+day_idx*paras]);
  if(diff<0){
  diff=-diff;
  }
  sum=sum+diff*(day_idx+1);//����ʱ��Ȩ��
}
score[day_idx+popu_idx*days]=sum;
}