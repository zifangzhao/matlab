
#include <hip/hip_runtime.h>
__global__ void stock_accelerate_profit(double * f,double * profit,double * p,int levels,int size1,int size2,int len_p){

int day_idx=threadIdx.x; //day idx
int para_idx=blockIdx.x; //parameter idx different para_estim
int para_idx2=size1;//threadIdx.y; //parameter_calculate data section
int popu_idx=blockIdx.y;

double post=0;

//f[paras+1,paras,day,popu],profit[days,levels],p[paras*levels,popu]
for(int idx=0;idx<levels;idx++){
post=post+profit[day_idx+(size2)*idx]*p[idx+(para_idx+popu_idx*size1)*levels];
}
f[para_idx2+(para_idx+(day_idx+popu_idx*(size2))*size1)*(size1+1)]=post;

//f[para_idx=profit[day_idx+blockDim.x*para_idx]*p[

}