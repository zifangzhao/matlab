
#include <hip/hip_runtime.h>
__global__ void stock_accelerate(double * f,double * data,double * p,int fft_day,int size1,int size2,int len_p){
//�������fΪpost���ֵ��Ϊ[paras1,paras2,days,popu]��������ÿ���ÿ�����͵�����Ԥ��ֵΪparas1���
int day_idx=threadIdx.x; //day idx
int para_idx=blockIdx.x; //parameter idx different para_estim
int para_idx2=threadIdx.y; //parameter_calculate data section
int popu_idx=blockIdx.y;
double temp=0;
double sum=0;
double post=0;


for(int idx=0;idx<fft_day;idx++){
    temp=data[para_idx2+(idx+day_idx)*size1];
    post=post+temp*p[(len_p/size1)*para_idx+idx*size1+para_idx2+popu_idx*len_p];
    sum=sum+temp;
}
post=post+(sum/fft_day)*p[(len_p/size1)*para_idx+fft_day*size1+para_idx2+popu_idx*len_p];
f[para_idx2+(para_idx+(day_idx+popu_idx*(size2-fft_day))*size1)*size1]=post;
//f[0]=1;
//f[1]=2;
//f[2]=3;

//f[para_idx2+para_idx*size1+day_idx*size1*size1]=fabs(post-data[para_idx+(day_idx+fft_day+1)
//*size1]);

}